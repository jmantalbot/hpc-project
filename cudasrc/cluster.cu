#include "hip/hip_runtime.h"
/*
Serial implementation based on the provided example/tutorial at
https://github.com/robertmartin8/RandomWalks/blob/master/kmeans.cpp
*/

#include <vector>
#include <float.h>
#include <cmath>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <iostream>
#include "point.hpp"
#include <iomanip>
#include "cluster.hpp"
#define BLOCK_SIZE 256



__global__
void computeDistances(float* coordinates, float* centroids, int* clusters, float* minDistances, int num_points, int k, int d, int* changed_count) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx >= num_points) return;
   float min_dist = FLT_MAX;
   int min_cluster = clusters[idx];
   for (int i = 0; i < k; i++){
      float dist = 0.0;
      for (int dim = 0; dim < d; dim++) {
         float diff = coordinates[idx * d + dim] - centroids[i * d + dim];
         dist += diff * diff;
      }
      dist = sqrtf(dist);
      if (dist < min_dist) {
         min_dist = dist;
         min_cluster = i;
      }
   }
   if(min_dist < minDistances[idx]) {
       minDistances[idx] = min_dist;
       if (clusters[idx] != min_cluster) {
           atomicAdd(changed_count, 1);
           clusters[idx] = min_cluster;
       }
   }
}


__global__
void computeSums(float* coordinates, int* clusters, float* sums, int* counts, int num_points, int k, int d) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if(idx >= num_points) return;
   int cluster = clusters[idx];
   if (cluster < 0  || cluster >= k) return;
   atomicAdd(&counts[cluster],1);
   for (int dim = 0; dim < d; dim++) {
      atomicAdd(&sums[cluster * d + dim], coordinates[idx * d + dim]);
   }
}


__global__
void updateCentroids(float* centroids, float* sums, int* counts, int k, int d) {
   int cluster = blockIdx.x;
   int dim = threadIdx.x;
   if (cluster >= k || dim >= d) return;
   int count = counts[cluster];
   if (count > 0) {
       centroids[cluster * d + dim] = sums[cluster * d + dim] / count;
   }
}


/* --- kMeansCluster ----
 * Determine the clusters for the given data points
 * Args:
 *   std::vector<Point>* points // in and out
 *   int maxEpochs // in
 *   int k // in
 */
void kMeansCluster(std::vector<Point>* points, int maxEpochs, int k){
    //bounds checking
    if (points->empty() || k <= 0 || maxEpochs <= 0) return;
    size_t num_points = points->size();
    size_t d = points->at(0).coordinates.size();
    float* h_coordinates = new float[num_points * d];
    int* h_clusters = new int[num_points];
    float* h_minDistances = new float[num_points];
    for(size_t i = 0; i < num_points; i++) {
        for (size_t j = 0; j < d; j++) {
            h_coordinates[i * d + j] = points->at(i).coordinates[j];
        }
        h_clusters[i] = -1;
        h_minDistances[i] = FLT_MAX;
    }
    float* h_centroids = new float[k * d];
    std::srand(100);
    for (int i = 0; i < k; i++) {
        int rand_idx = rand() % num_points;
        for (size_t j = 0; j < d; j++) {
            h_centroids[i * d + j] = h_coordinates[rand_idx * d + j];
        }
    }
    float *d_coordinates, *d_centroids, *d_minDistances, *d_sums;
    int *d_clusters, *d_counts, *d_changed;

    hipMalloc(&d_coordinates, num_points * d * sizeof(float));
    hipMalloc(&d_centroids, k * d * sizeof(float));
    hipMalloc(&d_clusters, num_points * sizeof(int));
    hipMalloc(&d_minDistances, num_points * sizeof(float));
    hipMalloc(&d_counts, k * sizeof(int));
    hipMalloc(&d_sums, k * d * sizeof(float));
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_coordinates, h_coordinates, num_points * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids, k * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusters, h_clusters, num_points * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minDistances, h_minDistances, num_points * sizeof(float), hipMemcpyHostToDevice);

    const int blockSize = BLOCK_SIZE;
    const int gridSize = (num_points + blockSize - 1) / blockSize;
    int h_changed = -1;
    for (int epoch = 0; epoch < maxEpochs; epoch++) {
        std::cout << "OMP EPOCH " << epoch << " Centroids:\n";
        for (int i = 0; i < k; i++) {
                 std::cout << "  Centroid " << i << ": (";
                 for (size_t dim = 0; dim < d; dim++) {
                     std::cout << std::fixed << std::setprecision(6) << h_centroids[i * d + dim];
                     if (dim < d - 1) std::cout << ", ";
                 }
                 std::cout <<")\n";
        }
        std::cout<<std::endl;
        if (h_changed == 0) {
            std::cout << "This Algorithm ran " << epoch << " times." << std::endl;
            break;
        }
        hipMemset(d_changed, 0, sizeof(int));
        computeDistances<<<gridSize, blockSize>>>(d_coordinates, d_centroids, d_clusters, d_minDistances, num_points, k, d, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(d_sums, 0, k * d * sizeof(float));
        hipMemset(d_counts, 0, k * sizeof(int));

        computeSums<<<gridSize, blockSize>>>(d_coordinates, d_clusters, d_sums, d_counts, num_points, k, d);
        hipDeviceSynchronize();
        updateCentroids<<<k,d>>>(d_centroids, d_sums, d_counts, k, d);
        hipDeviceSynchronize();
        hipMemcpy(h_centroids, d_centroids, k * d * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipMemcpy(h_clusters, d_clusters, num_points * sizeof(int), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < num_points; i++){
        points->at(i).cluster = h_clusters[i];
    }
        for (int i = 0; i < k; i++) {
                 std::cout << "  Centroid " << i << ": (";
                 for (size_t dim = 0; dim < d; dim++) {
                     std::cout << std::fixed << std::setprecision(6) << h_centroids[i * d + dim];
                     if (dim < d - 1) std::cout << ", ";
                 }
                 std::cout <<")\n";
        }
        std::cout<<std::endl;


    delete[] h_coordinates;
    delete[] h_clusters;
    delete[] h_minDistances;
    delete[] h_centroids;
    hipFree(d_coordinates);
    hipFree(d_centroids);
    hipFree(d_clusters);
    hipFree(d_minDistances);
    hipFree(d_counts);
    hipFree(d_sums);
    hipFree(d_changed);
}
