#include "hip/hip_runtime.h"
#include <vector>
#include <float.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "point.hpp"
#include "cluster.hpp"
#define BLOCK_SIZE 256

__global__ void computeLocalPointCenters(Point **localPoints, bool* localChanged, float *centroidCoordinates, int clusterId, int dimensions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Point point = *localPoints[idx];
    float distance = 0.0;
    for(size_t i = 0; i < dimensions; i++){
        distance += std::pow(centroidCoordinates[i] - *(point.coordinates + i), 2);
    }
    if (distance < point.minDistance) {
        //update the point's centroid (what cluster it belongs to)
        point.minDistance = distance;
        point.cluster = clusterId;
        *localChanged = true;
    }
}

void updateLocalPoints(
    std::vector<Point>* localPoints,
    std::vector<Point>* centroids,
    int numberOfCoordinates,
    int maxEpochs,
    int k
) {
    
    size_t num_points = localPoints->size();
    const int blockSize = BLOCK_SIZE;
    const int gridSize = (num_points + blockSize - 1) / blockSize;
    
    for (std::vector<Point>::iterator centroidIterator = centroids->begin(); centroidIterator != centroids->end(); centroidIterator++) {
            int clusterId = centroidIterator - centroids->begin();

            // The following is replaced by CUDA, which follows

            // for (std::vector<Point>::iterator pointIterator = localPoints->begin(); pointIterator != localPoints->end(); pointIterator++) {
            //     Point point = *pointIterator;
            //     double distance = centroidIterator->distance(point);
            //     if (distance < point.minDistance) {
            //         //update the point's centroid (what cluster it belongs to)
            //         point.minDistance = distance;
            //         point.cluster = clusterId;
            //         localChanged = true;
            //     }
            //     *pointIterator = point;
            //}

            Point *d_localpoints;
            bool *d_localchanged;

            hipMalloc(&d_localpoints, (size_t) (localPoints->end() - localPoints->begin()));
            hipMalloc(&d_localchanged, sizeof(bool));

            hipMemcpy(d_localpoints, localPoints->data(), localPoints->end() - localPoints->begin(), hipMemcpyHostToDevice);
            hipMemset(d_localchanged, false, sizeof(bool));

            computeLocalPointCenters<<<gridSize, blockSize>>>(&d_localpoints, d_localchanged, &centroidIterator->coordinates[0], clusterId, centroidIterator->coordinates.size());

            hipDeviceSynchronize();
        }
}