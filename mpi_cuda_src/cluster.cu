#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "cluster.h"

#define BLOCK_SIZE 256

__device__ float distance(const float* a, const float* b, int D) {
    float dist = 0.0f;
    for (int i = 0; i < D; ++i) {
        float diff = a[i] - b[i];
        dist += diff * diff;
    }
    return sqrtf(dist);
}

__global__ void computeAssignments(
    const float* coordinates,   // [num_points * D]
    const float* centroids,     // [k * D]
    int* assignments,           // [num_points]
    float* minDistances,        // [num_points]
    int* changed_count,
    int num_points,
    int k,
    int D
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    const float* point = &coordinates[idx * D];
    float min_dist = FLT_MAX;
    int min_cluster = -1;

    for (int i = 0; i < k; ++i) {
        const float* centroid = &centroids[i * D];
        float dist = distance(point, centroid, D);
        if (dist < min_dist) {
            min_dist = dist;
            min_cluster = i;
        }
    }

    if (assignments[idx] != min_cluster) {
        atomicAdd(changed_count, 1);
        assignments[idx] = min_cluster;
    }

    minDistances[idx] = min_dist;
}

__global__ void accumulateCentroids(
    const float* coordinates,
    const int* assignments,
    float* sums,
    int* counts,
    int num_points,
    int k,
    int D
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    int cluster = assignments[idx];
    if (cluster < 0 || cluster >= k) return;

    for (int d = 0; d < D; ++d) {
        atomicAdd(&sums[cluster * D + d], coordinates[idx * D + d]);
    }
    atomicAdd(&counts[cluster], 1);
}

__global__ void updateCentroids(
    float* centroids,
    const float* sums,
    const int* counts,
    int k,
    int D
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= k) return;

    int count = counts[i];
    if (count == 0) return;

    for (int d = 0; d < D; ++d) {
        centroids[i * D + d] = sums[i * D + d] / count;
    }
}

void run_kmeans_gpu(float* coords, int* assignments, float* centroids, int num_points, int k, int maxEpochs, int D) {
    float *d_coords, *d_centroids, *d_sums, *d_minDistances;
    int *d_assignments, *d_counts, *d_changed;

    size_t coord_size = num_points * D * sizeof(float);
    size_t centroid_size = k * D * sizeof(float);

    hipMalloc(&d_coords, coord_size);
    hipMalloc(&d_centroids, centroid_size);
    hipMalloc(&d_assignments, num_points * sizeof(int));
    hipMalloc(&d_minDistances, num_points * sizeof(float));
    hipMalloc(&d_counts, k * sizeof(int));
    hipMalloc(&d_sums, centroid_size);
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_coords, coords, coord_size, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, centroid_size, hipMemcpyHostToDevice);
    hipMemset(d_assignments, -1, num_points * sizeof(int));

    int h_changed = 1;
    int threads = BLOCK_SIZE;
    int blocks_points = (num_points + threads - 1) / threads;
    int blocks_centroids = (k + threads - 1) / threads;

    for (int epoch = 0; epoch < maxEpochs && h_changed > 0; epoch++) {
        hipMemset(d_changed, 0, sizeof(int));

        computeAssignments<<<blocks_points, threads>>>(
            d_coords, d_centroids, d_assignments, d_minDistances, d_changed,
            num_points, k, D);
        hipDeviceSynchronize();

        hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);

        hipMemset(d_sums, 0, centroid_size);
        hipMemset(d_counts, 0, k * sizeof(int));

        accumulateCentroids<<<blocks_points, threads>>>(
            d_coords, d_assignments, d_sums, d_counts,
            num_points, k, D);
        hipDeviceSynchronize();

        updateCentroids<<<blocks_centroids, threads>>>(
            d_centroids, d_sums, d_counts, k, D);
        hipDeviceSynchronize();
    }

    hipMemcpy(assignments, d_assignments, num_points * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(centroids, d_centroids, centroid_size, hipMemcpyDeviceToHost);

    hipFree(d_coords);
    hipFree(d_centroids);
    hipFree(d_assignments);
    hipFree(d_minDistances);
    hipFree(d_counts);
    hipFree(d_sums);
    hipFree(d_changed);
}
