#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "cluster.h"

#define D 3  // x, y, z
#define BLOCK_SIZE 256

struct point {
    int line_number;
    float x;
    float y;
    float z;
};

// Compute Euclidean distance between point and centroid
__device__ float distance(float* coords1, float* coords2) {
    float dx = coords1[0] - coords2[0];
    float dy = coords1[1] - coords2[1];
    float dz = coords1[2] - coords2[2];
    return sqrtf(dx * dx + dy * dy + dz * dz);
}

// Assign points to the closest centroid
__global__ void computeAssignments(
    float* coordinates,       // [num_points * 3]
    float* centroids,         // [k * 3]
    int* assignments,         // [num_points]
    float* minDistances,      // [num_points]
    int* changed_count,       // device-side counter
    int num_points,
    int k
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    float min_dist = FLT_MAX;
    int min_cluster = assignments[idx];

    float point[3] = {
        coordinates[idx * D],
        coordinates[idx * D + 1],
        coordinates[idx * D + 2]
    };

    for (int i = 0; i < k; i++) {
        float* centroid = &centroids[i * D];
        float dist = distance(point, centroid);
        if (dist < min_dist) {
            min_dist = dist;
            min_cluster = i;
        }
    }

    if (min_dist < minDistances[idx]) {
        minDistances[idx] = min_dist;
        if (assignments[idx] != min_cluster) {
            atomicAdd(changed_count, 1);
            assignments[idx] = min_cluster;
        }
    }
}

// Accumulate new centroid sums
__global__ void accumulateCentroids(
    float* coordinates,  // [num_points * 3]
    int* assignments,    // [num_points]
    float* sums,         // [k * 3]
    int* counts,         // [k]
    int num_points,
    int k
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    int cluster = assignments[idx];
    if (cluster < 0 || cluster >= k) return;

    atomicAdd(&sums[cluster * D + 0], coordinates[idx * D + 0]);
    atomicAdd(&sums[cluster * D + 1], coordinates[idx * D + 1]);
    atomicAdd(&sums[cluster * D + 2], coordinates[idx * D + 2]);
    atomicAdd(&counts[cluster], 1);
}

// Compute new centroids
__global__ void updateCentroids(
    float* centroids,  // [k * 3]
    float* sums,       // [k * 3]
    int* counts,       // [k]
    int k
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= k) return;

    int count = counts[i];
    if (count > 0) {
        centroids[i * D + 0] = sums[i * D + 0] / count;
        centroids[i * D + 1] = sums[i * D + 1] / count;
        centroids[i * D + 2] = sums[i * D + 2] / count;
    }
}

void run_kmeans_gpu(float* coords, int* assignments, float* centroids, int num_points, int k, int maxEpochs) {
    float *d_coords, *d_centroids, *d_sums, *d_minDistances;
    int *d_assignments, *d_counts, *d_changed;

    size_t coord_size = num_points * D * sizeof(float);
    size_t centroid_size = k * D * sizeof(float);

    hipMalloc(&d_coords, coord_size);
    hipMalloc(&d_centroids, centroid_size);
    hipMalloc(&d_assignments, num_points * sizeof(int));
    hipMalloc(&d_minDistances, num_points * sizeof(float));
    hipMalloc(&d_counts, k * sizeof(int));
    hipMalloc(&d_sums, centroid_size);
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_coords, coords, coord_size, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, centroid_size, hipMemcpyHostToDevice);


    hipMemset(d_assignments, -1, num_points * sizeof(int));
    hipMemset(d_minDistances, 0x7F, num_points * sizeof(float)); // Set to FLT_MAX

    int h_changed = 1;
    int threads = BLOCK_SIZE;
    int blocks_points = (num_points + threads - 1) / threads;
    int blocks_centroids = (k + threads - 1) / threads;

    for (int epoch = 0; epoch < maxEpochs && h_changed > 0; epoch++) {
        hipMemset(d_changed, 0, sizeof(int));
        computeAssignments<<<blocks_points, threads>>>(d_coords, d_centroids, d_assignments, d_minDistances, d_changed, num_points, k);
        hipDeviceSynchronize();

        hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);

        hipMemset(d_sums, 0, centroid_size);
        hipMemset(d_counts, 0, k * sizeof(int));

        accumulateCentroids<<<blocks_points, threads>>>(d_coords, d_assignments, d_sums, d_counts, num_points, k);
        hipDeviceSynchronize();

        updateCentroids<<<blocks_centroids, 1>>>(d_centroids, d_sums, d_counts, k);
        hipDeviceSynchronize();
    }

    hipMemcpy(assignments, d_assignments, num_points * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(centroids, d_centroids, centroid_size, hipMemcpyDeviceToHost);

    hipFree(d_coords);
    hipFree(d_centroids);
    hipFree(d_assignments);
    hipFree(d_minDistances);
    hipFree(d_counts);
    hipFree(d_sums);
    hipFree(d_changed);
}
